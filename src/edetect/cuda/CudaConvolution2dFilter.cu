#include "hip/hip_runtime.h"
/** @file
 * @brief Definition of CudaConvolution2dFilter class.
 *
 * @author Jan Bobek
 */

#include "edetect.hxx"
#include "cuda/CudaConvolution2dFilter.hxx"
#include "cuda/CudaError.hxx"
#include "cuda/CudaImage.hxx"

/// The convolution kernel.
__constant__ float cKernel[(2 * CudaConvolution2dFilter::MAX_RADIUS + 1) * (2 * CudaConvolution2dFilter::MAX_RADIUS + 1)];

/**
 * @brief CUDA kernel performing 2D discrete convolution.
 *
 * @param[out] dst
 *   The destination image data.
 * @param[in] dstStride
 *   Size of the row stride in destination data.
 * @param[in] src
 *   The source image data.
 * @param[in] srcStride
 *   Size of the row stride in source data.
 * @param[in] rows
 *   Number of rows in the image.
 * @param[in] cols
 *   Number of columns in the image.
 * @param[in] r
 *   Radius of the kernel.
 */
__global__ void
convolve2d(
    unsigned char* dst,
    unsigned int dstStride,
    const unsigned char* src,
    unsigned int srcStride,
    unsigned int rows,
    unsigned int cols,
    unsigned int r
    )
{
    const size_t col =
        blockIdx.x * blockDim.x + threadIdx.x;
    const size_t row =
        blockIdx.y * blockDim.y + threadIdx.y;

    if( row < rows && col < cols )
    {
        const size_t colStart = (col < r
                                 ? r - col : 0);
        const size_t colEnd = (cols <= col + r
                               ? cols - col + r - 1
                               : 2 * r);

        const size_t rowStart = (row < r
                                 ? r - row : 0);
        const size_t rowEnd = (rows <= row + r
                               ? rows - row + r - 1
                               : 2 * r);

        const unsigned char* rowp = src
            + (row - r + rowStart) * srcStride
            + (col - r + colStart) * sizeof(float);
        const float* colp;

        size_t i, j;
        float x = 0.0f;

        for( i = 0; i < rowStart; ++i )
        {
            colp = (float*)rowp;
            for( j = 0; j < colStart; ++j )
                x += *colp * cKernel[i * (2 * r + 1) + j];
            for(; j < colEnd; ++j, ++colp )
                x += *colp * cKernel[i * (2 * r + 1) + j];
            for(; j <= 2 * r; ++j )
                x += *colp * cKernel[i * (2 * r + 1) + j];
        }
        for(; i < rowEnd; ++i, rowp += srcStride )
        {
            colp = (float*)rowp;
            for( j = 0; j < colStart; ++j )
                x += *colp * cKernel[i * (2 * r + 1) + j];
            for(; j < colEnd; ++j, ++colp )
                x += *colp * cKernel[i * (2 * r + 1) + j];
            for(; j <= 2 * r; ++j )
                x += *colp * cKernel[i * (2 * r + 1) + j];
        }
        for(; i <= 2 * r; ++i )
        {
            colp = (float*)rowp;
            for( j = 0; j < colStart; ++j )
                x += *colp * cKernel[i * (2 * r + 1) + j];
            for(; j < colEnd; ++j, ++colp )
                x += *colp * cKernel[i * (2 * r + 1) + j];
            for(; j <= 2 * r; ++j )
                x += *colp * cKernel[i * (2 * r + 1) + j];
        }

        float* dstp = (float*)(dst + row * dstStride)
            + col;
        *dstp = x;
    }
}

/*************************************************************************/
/* CudaConvolution2dFilter                                               */
/*************************************************************************/
CudaConvolution2dFilter::CudaConvolution2dFilter(
    const float* kernel,
    unsigned int radius
    )
: mKernel( NULL ),
  mRadius( 0 )
{
    setKernel( kernel, radius );
}

void
CudaConvolution2dFilter::process(
    CudaImage& image
    )
{
    switch( image.format() )
    {
    case CudaImage::FMT_GRAY_FLOAT32:
        break;

    default:
    case CudaImage::FMT_GRAY_UINT8:
    case CudaImage::FMT_RGB_UINT8:
    case CudaImage::FMT_RGB_FLOAT32:
        throw std::runtime_error(
            "CudaConvolution2dFilter: Unsupported image format" );
    }

    // 32 = warp size, 8 * 32 = 256 threads
    const dim3 threadsPerBlock(32, 8);
    const dim3 numBlocks(
        (image.columns() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (image.rows() + threadsPerBlock.y - 1) / threadsPerBlock.y );

    CudaImage newImage(
        image.rows(), image.columns(),
        CudaImage::FMT_GRAY_FLOAT32 );

    cudaCheckError(
        hipMemcpyToSymbol(HIP_SYMBOL(
            cKernel), mKernel, (2 * mRadius + 1) * (2 * mRadius + 1) * sizeof(*mKernel),
            0, hipMemcpyHostToDevice ) );

    convolve2d<<< numBlocks, threadsPerBlock >>>(
        (unsigned char*)newImage.data(), newImage.rowStride(),
        (unsigned char*)image.data(), image.rowStride(),
        image.rows(), image.columns(), mRadius );

    cudaCheckLastError( "2D-convolution kernel launch failed" );
    cudaMsgCheckError( hipDeviceSynchronize(), "2D-convolution kernel run failed" );

    image.swap( newImage );
}

void
CudaConvolution2dFilter::setKernel(
    const float* kernel,
    unsigned int radius
    )
{
    if( MAX_RADIUS < radius )
        throw std::runtime_error(
            "CudaConvolution2dFilter: Kernel too large" );

    mKernel = kernel;
    mRadius = radius;
}
