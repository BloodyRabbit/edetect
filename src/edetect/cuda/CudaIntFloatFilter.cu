#include "hip/hip_runtime.h"
/** @file
 * @brief Definition of CudaIntFloatFilter class.
 *
 * @author Jan Bobek
 */

#include "common.hxx"
#include "cuda/CudaError.hxx"
#include "cuda/CudaIntFloatFilter.hxx"

/**
 * @brief CUDA kernel converting integer-pixels to float-pixels.
 *
 * @param[out] dst
 *   The destination image data.
 * @param[in] dstStride
 *   Size of the row stride in destination data.
 * @param[in] src
 *   The source image data.
 * @param[in] srcStride
 *   Size of the row stride in source data.
 * @param[in] rows
 *   Number of rows in the image.
 * @param[in] cols
 *   Number of columns in the image.
 */
__global__ void
convertInt2Float(
    unsigned char* dst,
    size_t dstStride,
    const unsigned char* src,
    size_t srcStride,
    size_t rows,
    size_t cols
    )
{
    const size_t col =
        blockIdx.x * blockDim.x + threadIdx.x;
    const size_t row =
        blockIdx.y * blockDim.y + threadIdx.y;

    if( col < cols && row < rows )
    {
        float* const dstp =
            (float*)(dst + row * dstStride) + col;
        const unsigned char* const srcp =
            src + row * srcStride + col;

        *dstp = *srcp / 255.0f;
    }
}

/**
 * @brief CUDA kernel converting float-pixels to integer-pixels.
 *
 * @param[out] dst
 *   The destination image data.
 * @param[in] dstStride
 *   Size of the row stride in destination data.
 * @param[in] src
 *   The source image data.
 * @param[in] srcStride
 *   Size of the row stride in source data.
 * @param[in] rows
 *   Number of rows in the image.
 * @param[in] cols
 *   Number of columns in the image.
 */
__global__ void
convertFloat2Int(
    unsigned char* dst,
    size_t dstStride,
    const unsigned char* src,
    size_t srcStride,
    size_t rows,
    size_t cols
    )
{
    const size_t col =
        blockIdx.x * blockDim.x + threadIdx.x;
    const size_t row =
        blockIdx.y * blockDim.y + threadIdx.y;

    if( col < cols && row < rows )
    {
        unsigned char* const dstp =
            dst + row * dstStride + col;
        const float* const srcp =
            (const float*)(src + row * srcStride) + col;

        *dstp = (unsigned char)(__saturatef(*srcp) * 255.0f);
    }
}

/*************************************************************************/
/* CudaIntFloatFilter                                                    */
/*************************************************************************/
const CudaImage::Format
CudaIntFloatFilter::FMT_TARGET[] =
{
    CudaImage::FMT_INVALID,      // FMT_INVALID
    CudaImage::FMT_GRAY_FLOAT32, // FMT_GRAY_UINT8
    CudaImage::FMT_GRAY_UINT8,   // FMT_GRAY_FLOAT32
    CudaImage::FMT_RGB_FLOAT32,  // FMT_RGB_UINT8
    CudaImage::FMT_RGB_UINT8,    // FMT_RGB_FLOAT32
};

void
CudaIntFloatFilter::process(
    CudaImage& image
    )
{
    const CudaImage::Format fmtTarget =
        FMT_TARGET[image.format()];

    // 32 = warp size, 8 * 32 = 256 threads
    const dim3 threadsPerBlock(32, 8);
    const dim3 numBlocks(
        (image.columns() * image.channels() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (image.rows() + threadsPerBlock.y - 1) / threadsPerBlock.y );

    CudaImage newImage(
        image.rows(), image.columns(), fmtTarget );

    switch( fmtTarget )
    {
    case CudaImage::FMT_RGB_FLOAT32:
    case CudaImage::FMT_GRAY_FLOAT32:
        convertInt2Float<<< numBlocks, threadsPerBlock >>>(
            (unsigned char*)newImage.data(), newImage.rowStride(),
            (unsigned char*)image.data(), image.rowStride(),
            image.rows(), image.columns() * image.channels() );
        break;

    case CudaImage::FMT_RGB_UINT8:
    case CudaImage::FMT_GRAY_UINT8:
        convertFloat2Int<<< numBlocks, threadsPerBlock >>>(
            (unsigned char*)newImage.data(), newImage.rowStride(),
            (unsigned char*)image.data(), image.rowStride(),
            image.rows(), image.columns() * image.channels() );
        break;

    default:
    case CudaImage::FMT_INVALID:
        throw std::runtime_error(
            "CudaIntFloatFilter: invalid format" );
    }

    cudaCheckLastError( "Int-Float conversion kernel launch failed" );
    cudaMsgCheckError( hipDeviceSynchronize(), "Int-Float conversion kernel run failed" );

    image.swap( newImage );
}
