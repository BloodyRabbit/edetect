#include "hip/hip_runtime.h"
/** @file
 * @brief Definition of CudaDesaturateFilter class.
 *
 * @author Jan Bobek
 */

#include "common.hxx"
#include "CudaDesaturateFilter.hxx"
#include "CudaError.hxx"
#include "CudaImage.hxx"

/**
 * @brief CUDA kernel for desaturation using
 *   the Average method.
 *
 * @param[out] dst
 *   The destination image data.
 * @param[in] dstStride
 *   Size of the row stride in destination data.
 * @param[in] src
 *   The source image data.
 * @param[in] srcStride
 *   Size of the row stride in source data.
 * @param[in] rows
 *   Number of rows in the image.
 * @param[in] cols
 *   Number of columns in the image.
 */
__global__ void
desaturateAverage(
    unsigned char* dst,
    size_t dstStride,
    const unsigned char* src,
    size_t srcStride,
    size_t rows,
    size_t cols
    )
{
    const size_t col =
        blockIdx.x * blockDim.x + threadIdx.x;
    const size_t row =
        blockIdx.y * blockDim.y + threadIdx.y;

    if( row < rows && col < cols )
    {
        float* const dstp =
            (float*)(dst + row * dstStride) + col;
        const float3* const srcp =
            (const float3*)(src + row * srcStride) + col;

        *dstp = (srcp->x + srcp->y + srcp->z) / 3.0f;
    }
}

/**
 * @brief CUDA kernel for desaturation using
 *   the Lightness method.
 *
 * @param[out] dst
 *   The destination image data.
 * @param[in] dstStride
 *   Size of the row stride in destination data.
 * @param[in] src
 *   The source image data.
 * @param[in] srcStride
 *   Size of the row stride in source data.
 * @param[in] rows
 *   Number of rows in the image.
 * @param[in] cols
 *   Number of columns in the image.
 */
__global__ void
desaturateLightness(
    unsigned char* dst,
    size_t dstStride,
    const unsigned char* src,
    size_t srcStride,
    size_t rows,
    size_t cols
    )
{
    const size_t col =
        blockIdx.x * blockDim.x + threadIdx.x;
    const size_t row =
        blockIdx.y * blockDim.y + threadIdx.y;

    if( row < rows && col < cols )
    {
        float* const dstp =
            (float*)(dst + row * dstStride) + col;
        const float3* const srcp =
            (const float3*)(src + row * srcStride) + col;

        const float a = fminf( srcp->x, srcp->y );
        const float b = fmaxf( srcp->x, srcp->y );
        const float c = fminf( a, srcp->z );
        const float d = fmaxf( b, srcp->z );

        *dstp = 0.5f * (c + d);
    }
}

/**
 * @brief CUDA kernel for desaturation using
 *   the Luminosity method.
 *
 * @param[out] dst
 *   The destination image data.
 * @param[in] dstStride
 *   Size of the row stride in destination data.
 * @param[in] src
 *   The source image data.
 * @param[in] srcStride
 *   Size of the row stride in source data.
 * @param[in] rows
 *   Number of rows in the image.
 * @param[in] cols
 *   Number of columns in the image.
 */
__global__ void
desaturateLuminosity(
    unsigned char* dst,
    size_t dstStride,
    const unsigned char* src,
    size_t srcStride,
    size_t rows,
    size_t cols
    )
{
    const size_t col =
        blockIdx.x * blockDim.x + threadIdx.x;
    const size_t row =
        blockIdx.y * blockDim.y + threadIdx.y;

    if( row < rows && col < cols )
    {
        float* const dstp =
            (float*)(dst + row * dstStride) + col;
        const float3* const srcp =
            (const float3*)(src + row * srcStride) + col;

        *dstp =
            /* z:RED y:GREEN x:BLUE */
            0.2126f * srcp->z
            + 0.7152f * srcp->y
            + 0.0722f * srcp->x;
    }
}

/*************************************************************************/
/* CudaDesaturateFilter                                                  */
/*************************************************************************/
CudaDesaturateFilter::CudaDesaturateFilter(
    CudaDesaturateFilter::Method method
    )
: mMethod( method )
{
}

void
CudaDesaturateFilter::process(
    CudaImage& image
    )
{
    switch( image.format() )
    {
    case CudaImage::FMT_GRAY_UINT8:
    case CudaImage::FMT_GRAY_FLOAT32:
        fputs( "CudaDesaturateFilter: Image already in grayscale\n", stderr );
        return;

    case CudaImage::FMT_RGB_FLOAT32:
        break;

    default:
    case CudaImage::FMT_RGB_UINT8:
        throw std::runtime_error(
            "CudaDesaturateFilter: Unsupported image format" );
    }

    // 32 = warp size, 8 * 32 = 256 threads
    const dim3 threadsPerBlock(32, 8);
    const dim3 numBlocks(
        (image.columns() + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (image.rows() + threadsPerBlock.y - 1) / threadsPerBlock.y );

    CudaImage newImage(
        image.rows(), image.columns(),
        CudaImage::FMT_GRAY_FLOAT32 );

    switch( mMethod )
    {
    case METHOD_AVERAGE:
        desaturateAverage<<< numBlocks, threadsPerBlock >>>(
            (unsigned char*)newImage.data(), newImage.rowStride(),
            (unsigned char*)image.data(), image.rowStride(),
            image.rows(), image.columns() );
        break;

    case METHOD_LIGHTNESS:
        desaturateLightness<<< numBlocks, threadsPerBlock >>>(
            (unsigned char*)newImage.data(), newImage.rowStride(),
            (unsigned char*)image.data(), image.rowStride(),
            image.rows(), image.columns() );
        break;

    case METHOD_LUMINOSITY:
        desaturateLuminosity<<< numBlocks, threadsPerBlock >>>(
            (unsigned char*)newImage.data(), newImage.rowStride(),
            (unsigned char*)image.data(), image.rowStride(),
            image.rows(), image.columns() );
        break;
    }

    cudaCheckLastError( "Desaturation kernel launch failed" );
    cudaMsgCheckError( hipDeviceSynchronize(), "Desaturation kernel run failed" );

    image.swap( newImage );
}
